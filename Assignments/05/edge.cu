
#include <sys/mman.h>

#include <iostream>
#include <fstream>
#include <span>
#include <sstream>
#include <string>
#include <iomanip>
#include <cstdlib>
#include <filesystem>

using Byte = unsigned char;

#define DEBUG 

#include "CudaCheck.h"
#include "Image.h"

#include "Kernels/Color.h"
#include "Kernels/greyscale.h"
#include "Kernels/convolve.h"
#include "Kernels/threshold.h"

int main(int argc, char* argv[]) {
    const char* Testfile = "Fox.ppm";
    const char* filename = argc > 1 ? argv[1] : Testfile;

    std::string baseName(filename);
    baseName = baseName.substr(0, baseName.length() - 4);  // Remove ".ppm"

    RGBImage img = readRGBImage(filename);

    const size_t width = img.width;
    const size_t height = img.height;
    const size_t numBytes = img.size();
    const size_t numPixels = img.numPixels(); 

    Color* gpuColor;
    CUDA_CHECK_CALL(hipMalloc(&gpuColor, numBytes));
    CUDA_CHECK_CALL(hipMemcpy(gpuColor, img.data(), numBytes, hipMemcpyHostToDevice));

    Byte* gpuGreyscale;
    CUDA_CHECK_CALL(hipMalloc(&gpuGreyscale, numPixels));

    dim3 blockDim(16, 16);
    dim3 numBlocks(width/blockDim.x + 1, height/blockDim.y + 1);
    greyscale<<<numBlocks, blockDim>>>(width, height, gpuColor, gpuGreyscale);

    Byte* gpuEdges;
    CUDA_CHECK_CALL(hipMalloc(&gpuEdges, numPixels));
    convolve<<<numBlocks, blockDim>>>(width, height, gpuGreyscale, gpuEdges);

    const Byte defaultThresholdValue = 128;
    std::cout << "Enter an integer value (0 to 255) or press Enter for default (" 
	    << static_cast<int>(defaultThresholdValue) 
	    << "): ";
    std::string threshInput;
    std::getline(std::cin, threshInput);
    const Byte thresholdValue = threshInput.empty() ? defaultThresholdValue : static_cast<Byte>(std::stoi(threshInput) % 256);
    threshold<<<numBlocks, blockDim>>>(width, height, gpuEdges, gpuColor, thresholdValue);

    Color* results = new Color[numPixels];
    CUDA_CHECK_CALL(hipMemcpy(results, gpuColor, numBytes, hipMemcpyDeviceToHost));

    std::string outDir = "Output";
    std::filesystem::create_directory(outDir);

    std::ostringstream thresholdStream;
    thresholdStream << std::setw(3) << std::setfill('0') << static_cast<int>(thresholdValue);
    std::string formattedThreshold = thresholdStream.str();
    
    std::string outputFileName = outDir + "/" + baseName + "-t" + formattedThreshold + ".ppm";

    std::ofstream outstream(outputFileName);
    outstream << RGBImage(width, height, results);

    std::cout << "Output: " << outputFileName << std::endl;
}
